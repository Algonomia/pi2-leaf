
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#define BLOCK_SIZE 32

#define FILENAME "data.csv"

void matrix_mult(double *a,double *b, double *c, int m, int n, int k)
{
    double sum = 0;
    for (int row = 0; row<m; row++) {
        for (int col = 0; col<k; col++) {
            sum = 0;
            for(int i = 0; i < n; i++) {
                sum += a[row * n + i] * b[i * k + col];
            }
            c[row * k + col] = sum;
        }
    }
}

__global__ void gpu_nodiag_normalize(double *A, double *I, int n, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n)
        if (x == i && x!=y){
            I[x*n + y] /= A[i*n + i];
            A[x*n + y] /= A[i*n + i];
        }

}

void nodiag_normalize(double *A, double *I, int n, int i){

    for (int x = 0; x<n; x++) {
        for (int y = 0; y<n; y++) {
            if (x == i && x!=y){
                I[x*n + y] /= A[i*n + i];
                A[x*n + y] /= A[i*n + i];
            }
        }
    }
}

__global__ void gpu_diag_normalize(double *A, double *I, int n, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n)
        if (x == y && x == i){
            I[x*n + y] /= A[i*n + i];
            A[x*n + y] /= A[i*n + i];
        }

}

void diag_normalize(double *A, double *I, int n, int i){

    for (int x = 0; x<n; x++) {
        for (int y = 0; y<n; y++) {
            if (x == y && x == i){
                I[x*n + y] /= A[i*n + i];
                A[x*n + y] /= A[i*n + i];
            }
        }
    }
}

__global__ void gpu_gaussjordan(double *A, double *I, int n, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n){
        if (x != i){
            I[x*n + y] -= I[i*n + y] * A[x*n + i];
            if (y != i){
                A[x*n + y] -= A[i*n + y] * A[x*n + i];
            }
        }
    }

}

void gaussjordan(double *A, double *I, int n, int i){

    for (int x = 0; x<n; x++) {
        for (int y = 0; y<n; y++) {
            if (x != i){
                I[x*n + y] -= I[i*n + y] * A[x*n + i];
                if (y != i){
                    A[x*n + y] -= A[i*n + y] * A[x*n + i];
                }
            }
        }
    }
}

__global__ void gpu_set_zero(double *A, double *I, int n, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n){
        if (x != i){
            if (y == i){
                A[x*n + y] = 0;
            }
        }
    }
}

void set_zero(double *A, double *I, int n, int i){

    for (int x = 0; x<n; x++) {
        for (int y = 0; y<n; y++) {
            if (x != i){
                if (y == i){
                    A[x*n + y] = 0;
                }
            }
        }
    }
}

void gpu_matrix_inv_gauss_jordan(double *d_A, double *result, int n)
{
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    for (int i = 0; i<n; i++) {
        gpu_nodiag_normalize<<<numBlocks, threadsPerBlock>>>(d_A, result, n, i);
        gpu_diag_normalize<<<numBlocks, threadsPerBlock>>>(d_A, result, n, i);
        gpu_gaussjordan<<<numBlocks, threadsPerBlock>>>(d_A, result, n, i);
        gpu_set_zero<<<numBlocks, threadsPerBlock>>>(d_A, result, n, i);
    }

}

void matrix_inv_gauss_jordan(double *d_A, double *result, int n)
{
    for (int i = 0; i<n; i++) {
        nodiag_normalize(d_A, result, n, i);
        diag_normalize(d_A, result, n, i);
        gaussjordan(d_A, result, n, i);
        set_zero(d_A, result, n, i);
    }

}

/*
*********************************************************************
function name: gpu_matrix_sum
description: sum of two matrix (only same size)
parameters:
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C)
            to store the result
Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/
__global__ void gpu_matrix_sum(double *a, double *b, double *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < n)
    {
        for(int i = 0; i < n; i++)
        {
            c[row * n + i] = a[row * n + i] + b[row * n + i];
        }
    }
}

/*
*********************************************************************
function name: gpu_matrix_diff
description: difference of two matrix (only same size)
parameters:
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C)
            to store the result
Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/
__global__ void gpu_matrix_diff(double *a, double *b, double *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < n)
    {
        for(int i = 0; i < n; i++)
        {
            c[row * n + i] = a[row * n + i] - b[row * n + i];
        }
    }
}

void matrix_diff(double *a, double *b, double *c, int n)
{
    for(int j = 0; j < n; j++)
    {
        for(int i = 0; i < n; i++)
        {
            double val = a[j * n + i] - b[j * n + i];
            c[j * n + i] = val;
        }
    }
}

/*
*********************************************************************
function name: gpu_matrix_identity
description: generate identity matrix
parameters:
            &a GPU device output purpose pointer to a n X n matrix (C)
            to store the result
Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/
__global__ void gpu_matrix_identity(double *a, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < n)
    {
        for(int i = 0; i < n; i++)
        {
            if(i == row) {
                a[row * n + i] = 1.0;
            } else {
                a[row * n + i] = 0.0;
            }
        }
    }
}

void matrix_identity(double *a, int n)
{
    for(int j = 0; j < n; j++)
    {
        for(int i = 0; i < n; i++)
        {
            if(i == j) {
                a[j * n + i] = 1.0;
            } else {
                a[j * n + i] = 0.0;
            }
        }
    }
}

/*
*********************************************************************
function name: gpu_matrix_diag
description: generate matrix by keeping only diagonal elements
parameters:
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device output purpose pointer to b n X n matrix (B)
            to store the result
Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/
__global__ void gpu_matrix_diag(double *a, double *b, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(row < n)
    {
        for(int i = 0; i < n; i++)
        {
            if(i == row) {
                b[row * n + i] = a[row * n + i];
            } else {
                b[row * n + i] = 0;
            }
        }
    }
}

void matrix_diag(double *a, double *b, int n)
{
    for(int j = 0; j < n; j++)
    {
        for(int i = 0; i < n; i++)
        {
            if(i == j) {
                b[j * n + i] = a[j * n + i];
            } else {
                b[j * n + i] = 0;
            }
        }
    }
}

/*
*********************************************************************
function name: cpu_matrix_mult
description: dot product of two matrix (not only square) in CPU,
             for validating GPU results
parameters:
            &a CPU device pointer to a n X n matrix (A)
            &b CPU device pointer to a n X n matrix (B)
            &c CPU device output purpose pointer to a n X n matrix (C)
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
return: none
*********************************************************************
*/
__global__ void gpu_square_matrix_mult(double *d_a, double *d_b, double *d_result, int n)
{
    __shared__ double tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    double tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

/*
*********************************************************************
function name: gpu_matrix_transpose
description: matrix transpose
parameters:
            &mat_in GPU device pointer to a rows X cols matrix
            &mat_out GPU device output purpose pointer to a cols X rows matrix
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
return: none
*********************************************************************
*/
__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}

/*
*********************************************************************
function name: gpu_matrix_transpose
description: matrix transpose
parameters:
            &mat_in GPU device pointer to a rows X cols matrix
            &mat_out GPU device output purpose pointer to a cols X rows matrix
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
return: none
*********************************************************************
*/
__global__ void gpu_matrix_norm(double *a, double *c, int m, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    double sum = 0;
    if(row < m)
    {
        for(int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * a[row * n + i];
        }
    }
    c[row] += sum;
}

int cpuComputation(FILE *fp, int n) {
    ssize_t line_size;
    char *line_buf = NULL;
    size_t line_buf_size = 0;
    int line_count = 0;

    /* Read matrix from file */
    double *h_a = static_cast<double *>(malloc(sizeof(double) * n * n));

    /* Loop through until we are done with the file. */
    do {
        int i = 0;

        /* Get the next line */
        line_size = getline(&line_buf, &line_buf_size, fp);

        /* Show the line details */
        char * pch;
        pch = strtok(line_buf, ",");
        while (pch != NULL && line_size >= 0)
        {
            h_a[line_count * n + i] = atof(pch);
            ++i;
            pch = strtok(NULL, ",");
        }

        /* Increment our line count */
        line_count++;
    } while (line_size >= 0);

    line_buf = NULL;

    /* Close the file now that we are done with it */
    fclose(fp);

    double *identity = static_cast<double *>(malloc(sizeof(double) * n * n));
    double *identityMinusA = static_cast<double *>(malloc(sizeof(double) * n * n));
    double *invertedMatrix = static_cast<double *>(malloc(sizeof(double) * n * n));
    double *diagMatrix = static_cast<double *>(malloc(sizeof(double) * n * n));
    double *invertedDiag = static_cast<double *>(malloc(sizeof(double) * n * n));
    double *productA = static_cast<double *>(malloc(sizeof(double) * n * n));
    double *productB = static_cast<double *>(malloc(sizeof(double) * n * n));

    fprintf(stdout, "Generating Identity Matrix\n");
    matrix_identity(invertedMatrix, n);
    matrix_identity(invertedDiag, n);
    matrix_identity(identity, n);

    fprintf(stdout, "Diff with adjacent Matrix\n");
    matrix_diff(identity, h_a, identityMinusA, n);

    fprintf(stdout, "Invert DiffMatrix\n");
    matrix_inv_gauss_jordan(identityMinusA, invertedMatrix, n);

    fprintf(stdout, "Get diagonal matrix from invertedMatrix\n");
    matrix_diag(invertedMatrix, diagMatrix, n);

    fprintf(stdout, "Invert diagonal matrix\n");
    matrix_inv_gauss_jordan(diagMatrix, invertedDiag, n);

    fprintf(stdout, "Multiply matrix\n");
    matrix_mult(invertedDiag, invertedMatrix, productA, n, n, n);
    matrix_mult(productA, h_a, productB, n, n, n);

    FILE *fpOut;
    fpOut = fopen("./output.csv","w");
    if (!fpOut) {
        fprintf(stderr, "Error opening file to write\n");
        return EXIT_FAILURE;
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            fprintf(fpOut,"%20.15f", productB[i*n + j]);
            if (j != n - 1) {
                fprintf(fpOut, ",");
            }
        }
        fprintf(fpOut, "\n");
    }

    fclose(fpOut);

    return EXIT_SUCCESS;
}

int cudaComputation(FILE *fp, int n) {
    char *line_buf = NULL;
    size_t line_buf_size = 0;
    ssize_t line_size;
    int line_count = 0;

    /* Read matrix from file */
    double *h_a, *h_b, *d_a;
    hipHostMalloc((void **) &h_a, sizeof(double)*n*n, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, sizeof(double)*n*n, hipHostMallocDefault);
    hipMalloc((void **) &d_a, sizeof(double)*n*n);

    fprintf(stdout, "Read FILE\n");
    /* Loop through until we are done with the file. */
    do {
        int i = 0;

        /* Get the next line */
        line_size = getline(&line_buf, &line_buf_size, fp);

        /* Show the line details */
        char * pch;
        pch = strtok(line_buf, ",");
        while (pch != NULL && line_size >= 0)
        {
            h_a[line_count * n + i] = atof(pch);
            ++i;
            pch = strtok(NULL, ",");
        }

        /* Increment our line count */
        line_count++;
    } while (line_size >= 0);

    line_buf = NULL;

    /* Close the file now that we are done with it */
    fclose(fp);

    // allocate memory in host RAM, h_cc is used to store CPU result
    hipMemcpy(d_a, h_a, sizeof(double)*n*n, hipMemcpyHostToDevice);

    double *identity, *identityMinusA, *invertedMatrix, *diagMatrix, *invertedDiag, *productA, *productB;
    hipHostMalloc((void **) &identity, sizeof(double)*n*n, hipHostMallocDefault);
    hipHostMalloc((void **) &identityMinusA, sizeof(double)*n*n, hipHostMallocDefault);
    hipHostMalloc((void **) &invertedMatrix, sizeof(double)*n*n, hipHostMallocDefault);
    hipHostMalloc((void **) &diagMatrix, sizeof(double)*n*n, hipHostMallocDefault);
    hipHostMalloc((void **) &invertedDiag, sizeof(double)*n*n, hipHostMallocDefault);
    hipHostMalloc((void **) &productA, sizeof(double)*n*n, hipHostMallocDefault);
    hipHostMalloc((void **) &productB, sizeof(double)*n*n, hipHostMallocDefault);

    hipMalloc((void **) &identity, sizeof(double)*n*n);
    hipMalloc((void **) &identityMinusA, sizeof(double)*n*n);
    hipMalloc((void **) &invertedMatrix, sizeof(double)*n*n);
    hipMalloc((void **) &diagMatrix, sizeof(double)*n*n);
    hipMalloc((void **) &invertedDiag, sizeof(double)*n*n);
    hipMalloc((void **) &productA, sizeof(double)*n*n);
    hipMalloc((void **) &productB, sizeof(double)*n*n);

    unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    fprintf(stdout, "Generating Identity Matrix\n");
    gpu_matrix_identity<<<dimGrid, dimBlock>>>(invertedMatrix, n);
    gpu_matrix_identity<<<dimGrid, dimBlock>>>(invertedDiag, n);
    gpu_matrix_identity<<<dimGrid, dimBlock>>>(identity, n);

    fprintf(stdout, "Diff with adjacent Matrix\n");
    gpu_matrix_diff<<<dimGrid, dimBlock>>>(identity, d_a, identityMinusA, n);

    fprintf(stdout, "Invert DiffMatrix\n");
    gpu_matrix_inv_gauss_jordan(identityMinusA, invertedMatrix, n);

    fprintf(stdout, "Get diagonal matrix from invertedMatrix\n");
    gpu_matrix_diag<<<dimGrid, dimBlock>>>(invertedMatrix, diagMatrix, n);

    fprintf(stdout, "Invert diagonal matrix\n");
    gpu_matrix_inv_gauss_jordan(diagMatrix, invertedDiag, n);

    fprintf(stdout, "Multiply matrix\n");
    gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(invertedDiag, invertedMatrix, productA, n);
    gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(productA, d_a, productB, n);

    hipMemcpy(h_b, productB, sizeof(double)*n*n, hipMemcpyDeviceToHost);

    FILE *fpOut;
    fpOut = fopen("./output.csv","w");
    if (!fpOut) {
        fprintf(stderr, "Error opening file to write\n");
        return EXIT_FAILURE;
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            fprintf(fpOut,"%20.15f", h_b[i*n + j]);
            if (j != n - 1) {
                fprintf(fpOut, ",");
            }
        }
        fprintf(fpOut, "\n");
    }

    fclose(fpOut);

    hipFree(d_a), hipFree(identity), hipFree(identityMinusA), hipFree(invertedMatrix),
            hipFree(diagMatrix), hipFree(invertedDiag), hipFree(productA), hipFree(productB);

    return EXIT_SUCCESS;
}


/*
*********************************************************************
function name: main
        description: test and compare
parameters:
none
return: none
*********************************************************************
*/
int main(int argc, char const *argv[])
{
    char fileName[100] = FILENAME;

    FILE *fp = fopen(fileName, "r");
    if (!fp) {
        fprintf(stderr, "Error opening file '%s'\n", fileName);
        return EXIT_FAILURE;
    }

    int n = 0;
    int ch;
    while(!feof(fp)) {
        ch = fgetc(fp);
        if(ch == '\n') {
            n++;
        }
    }
    rewind(fp);

    int devices = 0;

    hipError_t err = hipGetDeviceCount(&devices);

    if (devices > 0 && err == hipSuccess) {
        fprintf(stdout, "Launching CUDA Algo\n");
        return cudaComputation(fp, n);
    } else {
        fprintf(stdout, "Launching CPU Algo\n");
        return cpuComputation(fp, n);
    }

}
